#include "hip/hip_runtime.h"
#include "Flock.h"

VBOFlockResource::VBOFlockResource(): width(100), height(100)
{
	width = max(2, width);
	height = max(2, height);
	time = 0;
	generateGridFlock();
	
	hipError_t t = hipGraphicsGLRegisterBuffer(&cudaVBOFlock, bufferObject[VERTEX_BUFFER], cudaGraphicsMapFlagsNone); 
	if (t != hipSuccess)
	{ 
		printf("Failed with error: %s\n\n\n\n", hipGetErrorString(t));
	}
}

VBOFlockResource::~VBOFlockResource() {
	if (hipGraphicsUnregisterResource(cudaVBOFlock) != hipSuccess)
	{
			printf("Failed\n");
	}
}

void VBOFlockResource::generateGridFlock() {

	int loop_size = 2*height + 1;

	numVertices = width*height;
	numIndices = (width - 1)*loop_size;

	vertices = new Vector3[numVertices];
	normals = new Vector3[numVertices];
	textureCoords = new Vector2[numVertices];
	indices = new unsigned int [numIndices];

	type = GL_TRIANGLE_STRIP;
	for (int x = 0; x < width; x++) 
	{
		int loops = x*loop_size;
		for (int y = 0; y < height; y++)
		{
			int offset = y*width + x;

			if (x != width - 1)
				indices[loops + 2*y + 1] = offset;
			if (x != 0)
				indices[loops - loop_size + 2*y] = offset;

			vertices[offset] = Vector3(2*(x*1.0f/(width-1)) - 1, 0, 2*(y*1.0f/(height-1)) - 1);

			normals[offset] = Vector3(0,1,0);

			textureCoords[offset] = Vector2(x*1.0f/(width-1),y*1.0f/(height-1) );
		}
		if (x != width - 1)
		{
			indices[loops + loop_size - 1] = width*height;
		}
	}

	restart_index = width*height;

	BufferData();
	glBindVertexArray(0);
}

__global__ void vboFlockResource_update(float* ptr, int width, int height, float time) 
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int offset = y*width + x;
	if (x >= width || y >= height) return;

	float period = 10; // smaller number = fewer waves
	float rate = 1.0;  //smaller number = slower waves

	float cx = x*0.5f/width - 0.5f;//affects origin of waves ... probably
	float cy = y*0.5f/height - 0.5f;//affects origin of waves ... probably

	float wave = sin(sqrt(cx*cx + cy*cy)*period - rate*time);

	int sign = wave>0?1:-1;
	wave = sign*sqrt(sign*wave);

	ptr[3*offset + 1] = wave/20; //smaller number, more wavey waves

	period *= 3;
	rate *= -9;

	ptr[3*offset + 1] += (sin(x*period/(width - 1) + rate*time) + sin(y*period/(height - 1) + rate*time))/60;//bigger number,  more wavey waves
}

void VBOFlockResource::update(float msec) {
	 time += msec * 0.001f;//GameTimer().GetTimedMS();

	float* devBuff;
	size_t size;

	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks((width - 1)/threadsPerBlock.x + 1, (height - 1)/threadsPerBlock.y + 1);

	if (hipGraphicsMapResources(1, &cudaVBOFlock, 0) != hipSuccess)
	{
		printf("Failed\n");
	}

	hipGraphicsResourceGetMappedPointer((void**)&devBuff, &size, cudaVBOFlock);

	vboFlockResource_update<<<numBlocks, threadsPerBlock>>>(devBuff, width, height, time);

	if (hipGraphicsUnmapResources(1, &cudaVBOFlock, 0) != hipSuccess)
	{
		printf("Failed\n");
	}
}

void VBOFlockResource::initVBO(MeshBuffer type, float* data, int comps, int num, unsigned int mode) {
	glBindVertexArray(arrayObject);
	glGenBuffers(1, &bufferObject[type]);
	glBindBuffer(GL_ARRAY_BUFFER, bufferObject[type]);
	glBufferData(GL_ARRAY_BUFFER, num*comps*sizeof(GLfloat), (GLvoid*)data, mode);
	glVertexAttribPointer((GLuint)type, comps, GL_FLOAT, GL_FALSE, 0, 0);
	glEnableVertexAttribArray((GLuint)type);
	glBindVertexArray(0);
}

void VBOFlockResource::initIBO(unsigned int* data, int num, unsigned int mode) {
	glBindVertexArray(arrayObject);
	numIndices = num;
	glGenBuffers(1, &bufferObject[INDEX_BUFFER]);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, bufferObject[INDEX_BUFFER]);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, num * sizeof(GLuint), (GLvoid*)data, mode);
	glBindVertexArray(0);
}

void VBOFlockResource::draw() const {
	//glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	glPrimitiveRestartIndex(restart_index);
	glEnable(GL_PRIMITIVE_RESTART);
	{
	glBindVertexArray(arrayObject);
	glDrawElements(type, numIndices, GL_UNSIGNED_INT, 0);
	glBindVertexArray(0);
	}
	glDisable(GL_PRIMITIVE_RESTART);
	//glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
}